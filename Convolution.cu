#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>
#include <algorithm>
#include <assert.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct Startup{
    int threadsperblock = 1024;
} startup;

template<class T = int>
struct Result {
    float executiontime;
    std::vector<T> output;

    /*Move operator to only shallow copy vector*/
    Result& operator=(const Result& other) {
        executiontime = other.executiontime;
        output = std::move(other.output);
        return *this;
    }
};

template<class T = int>
struct KernelParameters {
    T* input;
    int  inputsize;
    T* filter;
    int  filtersize;
    T* output;
    int  outputsize;
    T  ghostvalue = 0;
};

template<class T = int>
__global__ void NaiveConvolution(KernelParameters<T> parameters){
    int outputindex = blockDim.x * blockIdx.x + threadIdx.x;

    if (outputindex < parameters.outputsize){
        T result = parameters.ghostvalue;
        int inputstart = outputindex - (parameters.filtersize/2)-1;
        for (int filterindex = 0; filterindex < parameters.filtersize; filterindex++) {
            int inputindex = inputstart + filterindex;
            if (inputindex >= 0 && inputindex < parameters.outputsize)
                result += parameters.input[inputindex] * parameters.filter[filterindex];
        }
        parameters.output[outputindex] = result;
    }
}

template<class T = int>
__global__ void ConstantConvolution(KernelParameters<T> parameters){
    
}

template<class T = int>
__global__ void SharedConvolution(KernelParameters<T> parameters){
    
}

template <class T>
using ConvolutionCudaKernel = void(*)(KernelParameters<T>);

template <class T>
const std::vector<ConvolutionCudaKernel<T>> cudaKernels{ 
    NaiveConvolution, ConstantConvolution, SharedConvolution 
};

bool isSymmetric(const std::vector<int>& vec){
    for(int i = 0; i < vec.size()/2; i++)
        if (vec[i] != vec[vec.size()-1-i])
            return false;
    return true;
}


int CalculateOutputSize(int inputsize, int filtersize){
    return inputsize+ filtersize-1;
}

/*
Result<T> CpuPerformConvolution(const std::vector<int>& input, const std::vector<int>& filter){
    std::vector<int> output(CalculateOutputSize(input.size(), filter.size()));

    bool isFilterSymmetric = isSymmetric(filter);
    for (int i = 0; i < input.size(); i++){
        if (isFilterSymmetric){
            for (int k = 0; k < filter.size()/2+1; k++){
                output[k+i] = filter[k] * input[k+i];
                output[(filter.size() - k - 1)+i] = filter[k] * input[(filter.size() - k - 1)+i];
            }
        }
    }

    Result r = {0, input};
    return std::move(r);
}*/

template<class T = int>
Result<T> CudaPerformConvolution(const std::vector<T>& input, const std::vector<T>& filter, ConvolutionCudaKernel<T> algorithm){
    T* device_input, *device_filter, *device_output; Result<T> result;
    std::vector<T> output(CalculateOutputSize(input.size(), filter.size()));

    gpuErrchk(hipMalloc((void **)&device_input,   input.size()*sizeof(T)));
    gpuErrchk(hipMalloc((void **)&device_filter, filter.size()*sizeof(T)));
    gpuErrchk(hipMalloc((void **)&device_output, output.size()*sizeof(T)));

    gpuErrchk(hipMemcpy(device_input,   input.data(),  input.size()*sizeof(T), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_filter, filter.data(), filter.size()*sizeof(T), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_output, output.data(), output.size()*sizeof(T), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    

    KernelParameters<T> parameters = { (T*)device_input, (int)input.size(), (T*)device_filter, (int)filter.size(), (T*)device_output, (int)output.size() };
    hipEventRecord(start);
    algorithm<<< output.size() / startup.threadsperblock+1, startup.threadsperblock>>>(parameters);
    gpuErrchk(hipEventRecord(stop));
    
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&result.executiontime, start, stop));

    gpuErrchk(hipMemcpy(output.data(), device_output, output.size()*sizeof(T), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(device_input)); gpuErrchk(hipFree(device_filter)); gpuErrchk(hipFree(device_output));

    result.output = std::move(output);
    return std::move(result);
}

/*Prints a few elements from the front and a few from the back*/
void printsome(const std::vector<int>& vec, int range){
    int rr = (vec.size()*2 > range) ? range : vec.size();
    int br = (vec.size()*2 > range) ? vec.size() - range/2 : vec.size();
    for (int i = 0; i < rr; i++)
        std::cout << vec[i] << ", ";
    std::cout << "... ";
    for (int i = br; i < vec.size(); i++)
        std::cout << vec[i] << ", ";
    std::cout << std::endl;
}

void printall(const std::vector<int>& vec) {
    for (auto e : vec)
        std::cout << e << ", ";
}

/*Tests the example in the lecture slides*/
void TestLectureExample(){
    //std::vector<int> input{1,4,2,5};
    //std::vector<int> filter{1,4,3};
    //Result<int> r = CudaPerformConvolution(input, filter, NaiveConvolution);
    //assert(std::equal(r.output.begin(), r.output.end(), std::vector<int>{ 3, 16, 23, 27, 22, 5 }.begin() ));

    std::vector<float> inputf{1.,4.,2.,5.};
    std::vector<float> filterf{1.,4.,3.};
    Result<float> r1 = CudaPerformConvolution(inputf, filterf, NaiveConvolution);
    assert(std::equal(r1.output.begin(), r1.output.end(), std::vector<float>{ 3., 16., 23., 27., 22., 5. }.begin() ));

}

int main(int argc, char** argv){

    TestLectureExample();

    int inputsize = 1024*1024;
    std::vector<int> input(inputsize);//(inputsize);
    std::generate(input.begin(), input.end(), []() { static int x = 0; x++;return x; });
    std::vector<int> filter(101);
    std::generate(filter.begin(), filter.end(), []() { static int x = -1; x++;return x; });

    //Result r = CpuPerformConvolution(input, filter);

    for (ConvolutionCudaKernel<int> cudakern : cudaKernels<int>){
        Result<int> r1 = CudaPerformConvolution(input, filter, cudakern);
        std::cout << "Kernel Executed in: " << r1.executiontime << " milliseconds" << std::endl;
        printsome(r1.output, 10);
        std::cout << std::endl << std::endl;
    }
}
