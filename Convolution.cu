#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>
#include <algorithm>
#include <assert.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct Startup{
    int threadsperblock = 1024;
} startup;

template<class T = int>
struct Result {
    float executiontime;
    std::vector<T> output;

    /*Move operator to only shallow copy vector*/
    Result& operator=(const Result& other) {
        executiontime = other.executiontime;
        output = std::move(other.output);
        return *this;
    }
};

template<class T = int>
struct KernelParameters {
    T* input;
    int  inputsize;
    T* filter;
    int  filtersize;
    T* output;
    int  outputsize;
    T  ghostvalue = (T)0;
};

template<class T = int>
__global__ void NaiveConvolution(KernelParameters<T> parameters){
    int outputindex = blockDim.x * blockIdx.x + threadIdx.x;

    if (outputindex < parameters.outputsize){
        T result = parameters.ghostvalue;
        int inputstart = outputindex - (parameters.filtersize/2)-1;
        for (int filterindex = 0; filterindex < parameters.filtersize; filterindex++) {
            int inputindex = inputstart + filterindex;
            if (inputindex >= 0 && inputindex < parameters.outputsize-2)
                result += parameters.input[inputindex] * parameters.filter[filterindex];
        }
        parameters.output[outputindex] = result;
    }
}

template<class T = int>
__global__ void ConstantConvolution(KernelParameters<T> parameters){
    
}

template<class T = int>
__global__ void SharedConvolution(KernelParameters<T> parameters){
    
}

template <class T>
using ConvolutionCudaKernel = void(*)(KernelParameters<T>);

template <typename T>
const std::vector<ConvolutionCudaKernel<T>> cudaKernels{ 
    NaiveConvolution<T>, ConstantConvolution<T>, SharedConvolution<T> 
};

bool isSymmetric(const std::vector<int>& vec){
    for(int i = 0; i < vec.size()/2; i++)
        if (vec[i] != vec[vec.size()-1-i])
            return false;
    return true;
}


int CalculateOutputSize(int inputsize, int filtersize){
    return inputsize+ filtersize-1;
}

/*
Result<T> CpuPerformConvolution(const std::vector<int>& input, const std::vector<int>& filter){
    std::vector<int> output(CalculateOutputSize(input.size(), filter.size()));

    bool isFilterSymmetric = isSymmetric(filter);
    for (int i = 0; i < input.size(); i++){
        if (isFilterSymmetric){
            for (int k = 0; k < filter.size()/2+1; k++){
                output[k+i] = filter[k] * input[k+i];
                output[(filter.size() - k - 1)+i] = filter[k] * input[(filter.size() - k - 1)+i];
            }
        }
    }

    Result r = {0, input};
    return std::move(r);
}*/

template<class T = int>
Result<T> CudaPerformConvolution(const std::vector<T>& input, const std::vector<T>& filter, ConvolutionCudaKernel<T> algorithm){
    T* device_input, *device_filter, *device_output; Result<T> result;
    std::vector<T> output(CalculateOutputSize(input.size(), filter.size()));

    gpuErrchk(hipMalloc((void **)&device_input,   input.size()*sizeof(T)));
    gpuErrchk(hipMalloc((void **)&device_filter, filter.size()*sizeof(T)));
    gpuErrchk(hipMalloc((void **)&device_output, output.size()*sizeof(T)));

    gpuErrchk(hipMemcpy(device_input,   input.data(),  input.size()*sizeof(T), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_filter, filter.data(), filter.size()*sizeof(T), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_output, output.data(), output.size()*sizeof(T), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    

    KernelParameters<T> parameters = { (T*)device_input, (int)input.size(), (T*)device_filter, (int)filter.size(), (T*)device_output, (int)output.size() };
    hipEventRecord(start);
    algorithm<<< output.size() / startup.threadsperblock+1, startup.threadsperblock>>>(parameters);
    gpuErrchk(hipEventRecord(stop));
    
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&result.executiontime, start, stop));

    gpuErrchk(hipMemcpy(output.data(), device_output, output.size()*sizeof(T), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(device_input)); gpuErrchk(hipFree(device_filter)); gpuErrchk(hipFree(device_output));

    result.output = std::move(output);
    return std::move(result);
}

/*Prints a few elements from the front and a few from the back*/
void printsome(const std::vector<int>& vec, int range){
    int rr = (vec.size()*2 > range) ? range : vec.size();
    int br = (vec.size()*2 > range) ? vec.size() - range/2 : vec.size();
    for (int i = 0; i < rr; i++)
        std::cout << vec[i] << ", ";
    std::cout << "... ";
    for (int i = br; i < vec.size(); i++)
        std::cout << vec[i] << ", ";
    std::cout << std::endl;
}

template<class T = int>
void printall(const std::vector<T>& vec) {
    for (auto e : vec)
        std::cout << e << ", ";
}

/*Tests the example in the lecture slides*/
void TestLectureExample(){
    /*Test Integer Convolution*/
    {
        std::vector<int> input{1,4,2,5};
        std::vector<int> filter{1,4,3};
        Result<int> r = CudaPerformConvolution(input, filter, NaiveConvolution);
        std::cout << "Testing: "; printall(input); std::cout << std::endl;
        std::cout << " Result: "; printall(r.output); std::cout << std::endl;
        assert(std::equal(r.output.begin(), r.output.end(), std::vector<int>{ 3, 16, 23, 27, 22, 5 }.begin() ));
    }

    /*Test Floating Point Convolution*/
    {
        std::vector<float> input{.5, 2., 1.,2.5};
        std::vector<float> filter{.5,.2,1.5};
        Result<float> r = CudaPerformConvolution(input, filter, NaiveConvolution);
        std::cout << "Testing: "; printall(input); std::cout << std::endl;
        std::cout << " Result: "; printall(r.output); std::cout << std::endl;
        assert(std::equal(r.output.begin(), r.output.end(), std::vector<float>{ 0.75, 3.1, 2.15, 4.95, 1, 1.25 }.begin() ));

    }
}

int main(int argc, char** argv){

    TestLectureExample();


    int inputsize = 1024*1024*256;
    std::vector<int> input(inputsize);//(inputsize);
    std::generate(input.begin(), input.end(), []() { static int x = 0; x++;return x; });
    std::vector<int> filter(300);
    std::generate(filter.begin(), filter.end(), []() { static int x = -1; x++;return x; });

    //Result r = CpuPerformConvolution(input, filter);
    
    //for (auto it = cudaKernels<int>.begin(); it != cudaKernels.end(); it++){
    //    Result<int> r1 = CudaPerformConvolution(input, filter, *it);
    //    std::cout << "Kernel Executed in: " << r1.executiontime << " milliseconds" << std::endl;
    //    printsome(r1.output, 10);
    //    std::cout << std::endl << std::endl;
    //}
    //for (auto cudakern : cudaKernels<int>){
    std::cout << std::endl;
    Result<int> r1 = CudaPerformConvolution(input, filter, NaiveConvolution);
    std::cout << "Kernel Executed in: " << r1.executiontime << " milliseconds" << std::endl;
    printsome(r1.output, 10);
    std::cout << std::endl << std::endl;
    //}
}
