#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>
#include <algorithm>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct Startup{
    int threadsperblock = 1024;
} startup;

struct Result {
    float executiontime;
    std::vector<int> output;

    /*Move operator to only shallow copy vector*/
    Result& operator=(const Result& other) {
        executiontime = other.executiontime;
        output = std::move(other.output);
        return *this;
    }
};

struct KernelParameters {
    int* input;
    int  inputsize;
    int* filter;
    int  filtersize;
    int* output;
    int  outputsize;
    int  ghostvalue = 0;
};

__global__ void NaiveConvolution(KernelParameters parameters){
    int outputindex = blockDim.x * blockIdx.x + threadIdx.x;

    if (outputindex < parameters.outputsize){
        int result = parameters.ghostvalue;
        int inputstart = outputindex - (parameters.filtersize/2)-1;
        for (int filterindex = 0; filterindex < parameters.filtersize; filterindex++) {
            int inputindex = inputstart + filterindex;
            if (inputindex >= 0 && inputindex < parameters.outputsize)
                result += parameters.input[inputindex] * parameters.filter[filterindex];
        }
        parameters.output[outputindex] = result;
    }
}

__global__ void ConstantConvolution(KernelParameters parameters){
    
}

__global__ void SharedConvolution(KernelParameters parameters){
    
}

typedef void(*ConvolutionCudaKernel)(KernelParameters);
const std::vector<ConvolutionCudaKernel> cudaKernels{ 
    NaiveConvolution, ConstantConvolution, SharedConvolution 
};

bool isSymmetric(const std::vector<int>& vec){
    for(int i = 0; i < vec.size()/2; i++)
        if (vec[i] != vec[vec.size()-1-i])
            return false;
    return true;
}


int CalculateOutputSize(int inputsize, int filtersize){
    return inputsize+ filtersize-1;
}

Result CpuPerformConvolution(const std::vector<int>& input, const std::vector<int>& filter){
    std::vector<int> output(CalculateOutputSize(input.size(), filter.size()));

    bool isFilterSymmetric = isSymmetric(filter);
    for (int i = 0; i < input.size(); i++){
        if (isFilterSymmetric){
            for (int k = 0; k < filter.size()/2+1; k++){
                output[k+i] = filter[k] * input[k+i];
                output[(filter.size() - k - 1)+i] = filter[k] * input[(filter.size() - k - 1)+i];
            }
        }
    }

    Result r = {0, input};
    return std::move(r);
}

Result CudaPerformConvolution(const std::vector<int>& input, const std::vector<int>& filter, ConvolutionCudaKernel algorithm){
    int* device_input, *device_filter, *device_output; Result result;
    std::vector<int> output(CalculateOutputSize(input.size(), filter.size()));

    gpuErrchk(hipMalloc((void **)&device_input,   input.size()*sizeof(int)));
    gpuErrchk(hipMalloc((void **)&device_filter, filter.size()*sizeof(int)));
    gpuErrchk(hipMalloc((void **)&device_output, output.size()*sizeof(int)));

    gpuErrchk(hipMemcpy(device_input,   input.data(),  input.size()*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_filter, filter.data(), filter.size()*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(device_output, output.data(), output.size()*sizeof(int), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    

    KernelParameters parameters = { (int*)device_input, (int)input.size(), (int*)device_filter, (int)filter.size(), (int*)device_output, (int)output.size() };
    hipEventRecord(start);
    algorithm<<< output.size() / startup.threadsperblock+1, startup.threadsperblock>>>(parameters);
    gpuErrchk(hipEventRecord(stop));
    
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&result.executiontime, start, stop));

    gpuErrchk(hipMemcpy(output.data(), device_output, output.size()*sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(device_input)); gpuErrchk(hipFree(device_filter)); gpuErrchk(hipFree(device_output));

    result.output = std::move(output);
    return std::move(result);
}

int main(int argc, char** argv){
    //int inputsize = 1024*1024;
    std::vector<int> input{1,4,2,5,235,234,1};//(inputsize);
    //std::generate(input.begin(), input.end(), []() { static int x = -1; x++;return x; });
    std::vector<int> filter{1,4,3,1};
    //std::generate(filter.begin(), filter.end(), []() { static int x = -1; x++;return x; });

    Result r = CpuPerformConvolution(input, filter);

    for (ConvolutionCudaKernel cudakern : cudaKernels){
        Result r1 = CudaPerformConvolution(input, filter, cudakern);
        std::cout << "Kernel Executed in: " << r1.executiontime << " milliseconds" << std::endl;
        //for (int i = 0; i < 10; i++)
        //    std::cout << r1.output[i] << ", ";
        //std::cout << "... ";
        //for (int i = r1.output.size()-5; i < r1.output.size(); i++)
        //    std::cout << r1.output[i] << ", ";
        for (auto e : r1.output){
            std::cout << e << ", ";
        }
        std::cout << std::endl << std::endl;
    }
}
